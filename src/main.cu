#include <iostream>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "Mesher.cuh"

int main() {

    int count;
    auto err = hipGetDeviceCount(&count);
    if(err != hipSuccess){

        std::cout << "Error: " << hipGetErrorName(err) << std::endl;
        throw std::runtime_error("cuda error");
    }

    auto msh = Mesh<MeshUtils::Dim::D1>(MeshUtils::Units::METERS, 100, -100, 100);

    return 0;
}
