#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "Mesher.cuh"
#include "HelmholtzSet.cuh"
#include <memory>
#include "cuda_types_utils/cuda_float3_operators.cuh"
#include "CudaAllocatorInterfaces.cuh"
#include "EvalSystemForMesh.cuh"

int main() {

    using namespace CUDAUtils;
    constexpr unsigned int size{30};

    auto coils = Memory::newCudaInstance<HelmholtzSet>(8, 60, 1.9, 200, SimulatorUtils::Geometry::Plane::XY, 100, 0.1);

    auto mesh = Memory::newCudaInstance<Mesh<MeshUtils::Dim::D3, size>>(MeshUtils::Units::CENTIMETERS,
                                                                        -100, 100,
                                                                        -100, 100,
                                                                        -100, 100);

    typedef float3 rarr[size][size];
    rarr *resultGPU;
    CUDA_ERRCHK(hipMalloc((void**)&resultGPU, sizeof(float3)*size*size*size)) // Allocating GPU memory for result.

    dim3 threads(size,size,1);
    dim3 blocks(size,1,1);

    EvalSystemForMesh<<<blocks, threads>>>(coils, mesh, resultGPU);

    float3 resultHost[size][size][size];
    CUDA_ERRCHK(hipMemcpy(resultHost, resultGPU, sizeof(float3)*size*size*size, hipMemcpyDeviceToHost))

    hipFree(resultGPU);
    Memory::deleteCudaInstance(mesh);
    Memory::deleteCudaInstance(coils);

    std::cout << resultHost[5][5][6].x << std::endl;

    return 0;
}
