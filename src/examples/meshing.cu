#include <iostream>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "Mesher.cuh"
#include <memory>
#include "cuda_float3_operators.cuh"

int main() {

    int count;
    auto err = hipGetDeviceCount(&count);
    if(err != hipSuccess){

        std::cout << "Error: " << hipGetErrorName(err) << std::endl;
        throw std::runtime_error("cuda error");
    }

    auto msh1D = std::make_unique<Mesh<MeshUtils::Dim::D1, 100>>(MeshUtils::Units::METERS, -100, 100);

    auto msh2D = std::make_unique<Mesh<MeshUtils::Dim::D2, 100>>(MeshUtils::Units::METERS, -100, 100, -100, 100);

    auto msh3D = std::make_unique<Mesh<MeshUtils::Dim::D3, 100>>(MeshUtils::Units::METERS, -100, 100, -100, 100, -100, 100);

    auto val = (*msh3D).get(99,99,99);

    std::cout << val << std::endl;
    auto sizeFrac = (float)Mesh<MeshUtils::Dim::D3, 100>::size() / (float)Mesh<MeshUtils::Dim::D3, 50>::size();
    std::cout << sizeFrac << std::endl;

    return 0;
}
