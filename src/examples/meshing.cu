#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "Mesher.cuh"
#include "cuda_float3_operators.cuh"
#include "hip/hip_vector_types.h"
#include "CudaMacros.cuh"

#define N 100

int main() {

    int count;
    auto err = hipGetDeviceCount(&count);
    if(err != hipSuccess){

        std::cout << "Error: " << hipGetErrorName(err) << std::endl;
        throw std::runtime_error("cuda error");
    }

    using MeshType = Mesh<MeshUtils::Dim::D3, N>;

    auto msh3D = new MeshType(MeshUtils::Units::METERS, -100, 100, -100, 100, -100, 100);

    auto val = msh3D->get(0, 18, 92);

    hipDeviceSetLimit(hipLimitMallocHeapSize, 256*1024*1024);

    std::cout << "CPU generated coordinates: " << val << std::endl;

    float3* gpuPoint;
    CUDA_ERRCHK(hipMalloc((void**)&gpuPoint, sizeof(float3)))

    msh3D->newCudaInstance();

    auto ptr = msh3D -> getCudaInstancePtr();

    getPointGPUMesh3D<<<1,1>>>(ptr, 0, 18, 92, gpuPoint);

    msh3D->deleteCudaInstance();

    float3 hostPoint;
    CUDA_ERRCHK(hipMemcpy(&hostPoint, gpuPoint, sizeof(float3), hipMemcpyDeviceToHost))

    std::cout << "The same GPU generated coordinates: " << hostPoint << std::endl;

    hipFree(gpuPoint);

    float3* gpuPoint2;
    CUDA_ERRCHK(hipMalloc((void**)&gpuPoint2, sizeof(float3)))

    auto test = CUDAUtils::Memory::newCudaInstance<MeshType>(MeshUtils::Units::METERS, -100, 100, -100, 100, -100, 100);

    getPointGPUMesh3D<<<1,1>>>(test, 0, 18, 92, gpuPoint2);

    float3 hostPoint2;
    CUDA_ERRCHK(hipMemcpy(&hostPoint2, gpuPoint2, sizeof(float3), hipMemcpyDeviceToHost))

    std::cout << "The same GPU generated coordinates #2: " << hostPoint << std::endl;

    deleteCudaInstance(test);

    delete msh3D;

    return 0;
}
