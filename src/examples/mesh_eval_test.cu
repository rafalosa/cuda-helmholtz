#include <iostream>
#include <memory>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "Mesher.cuh"
#include "cuda_float3_operators.cuh"
#include "hip/hip_vector_types.h"
#include "CudaMacros.cuh"
#include "HelmholtzSet.cuh"
#include "Utils.cuh"

using namespace CUDAUtils::Memory;

int main() {

    auto GPUCoils = newCudaInstance<HelmholtzSet>(8, .5, .0019, .74,
                                                                     SimulatorUtils::Geometry::Plane::XY, 100, 0.1);
    auto GPUMesh = newCudaInstance<Mesh<MeshUtils::Dim::D3, 10>>
    (MeshUtils::Units::CENTIMETERS, -100, 100, -100, 100, -100, 100);

    /// Calculations code....

    deleteCudaInstance(GPUMesh);
    deleteCudaInstance(GPUCoils);

    return 0;
}