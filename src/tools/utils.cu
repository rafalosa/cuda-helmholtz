#include "hip/hip_runtime.h"
//
// Created by rafal on 27.03.2022.
//

#include "utils.cuh"

namespace CUDAUtils{

    __host__ void showCudaDeviceProps(int device){

        hipDeviceProp_t props{};

        hipGetDeviceProperties(&props, device);

        std::cout << "--- Information for device "<< device <<" ---" << std::endl;
        std::cout << "Name: " << props.name << std::endl;
        std::cout << "Compute capability: " << props.major <<"."<<props.minor << std::endl;
        std::cout << "Clock rate: " << props.clockRate << std::endl;
        std::cout << "Copy overlap: " << (props.deviceOverlap ? "Enabled" : "Disabled") << std::endl;
        std::cout << "Kernel execution timeout: " << (props.kernelExecTimeoutEnabled ? "Enabled" : "Disabled") << std::endl;
        std::cout << "Total global memory: " << props.totalGlobalMem << std::endl;
        std::cout << "Total const. memory: " << props.totalConstMem << std::endl;
        std::cout << "Memory pitch: " << props.memPitch << std::endl;
        std::cout << "Multiprocessor (mp) count: " << props.multiProcessorCount << std::endl;
        std::cout << "Shared memory per mp: " << props.sharedMemPerMultiprocessor << std::endl;
        std::cout << "Max threads per block: " << props.maxThreadsPerBlock << std::endl;
    }
}

namespace SimulatorUtils {
    namespace Structures {
        float &vec3D::operator[](unsigned int ind) { return _vals[ind]; }

        vec3D::vec3D(std::initializer_list<float> list) {
            size_t i = 0;
            for (auto &val: list) {
                _vals[i++] = val;
            }
        }
    }
    namespace Math {
        __host__ __device__ float3 crossProduct(float3 v1, float3 v2) {
            return make_float3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
        }

        __host__ __device__ float norm(float3 vec) {

            return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
        }

        __host__ __device__ void assignLinearSpace(float boundary1, float boundary2, size_t steps, float* target, float multiplier = 1){

            auto linearSpan = (boundary2 - boundary1) * multiplier;
            float step = (linearSpan / (float)steps);

            for(size_t i=0; i <= steps; i++){

                target[i] = boundary1 * multiplier + (float)i * step;

            }
        }
    }
}